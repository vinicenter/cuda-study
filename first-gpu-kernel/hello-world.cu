
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printKernel() {
    printf("Hello from CUDA kernel! Block: %d, Thread: %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    printKernel<<<2, 4>>>();

    hipDeviceSynchronize();

    return 0;
}
